#include "hip/hip_runtime.h"
#include <png.h>
#include <stdio.h>
#include <stdlib.h>
#include <stdint.h>
#include <complex>
#include <iostream>
#include <cmath>
#include <thrust/complex.h>

using thrust::complex;
using thrust::exp;
using thrust::norm;

// A coloured pixel.
struct Pixel {
  uint8_t red;
  uint8_t green;
  uint8_t blue;
};

// A picture.
struct Bitmap {
  Pixel* pixels;
  size_t width;
  size_t height;
};

// Given "bitmap", this returns the pixel of bitmap at the point ("x", "y").
__host__ __device__
Pixel* pixel_at(Bitmap* bitmap, int x, int y) {
  return bitmap->pixels + bitmap->width * y + x;
}

/* Write "bitmap" to a PNG file specified by "path"; returns 0 on
   success, non-zero on error. */
int save_png_to_file(Bitmap* bitmap, const char* path) {
  FILE* fp;
  png_structp png_ptr = NULL;
  png_infop info_ptr = NULL;
  size_t x, y;
  png_byte** row_pointers = NULL;

  /* "status" contains the return value of this function. At first
     it is set to a value which means 'failure'. When the routine
     has finished its work, it is set to a value which means
     'success'. */
  int status = -1;

  /* The following number is set by trial and error only. I cannot
     see where it is documented in the libpng manual. */
  int pixel_size = 3;
  int depth = 8;

  fp = fopen(path, "wb");
  if (!fp) {
    return status;
  }

  png_ptr = png_create_write_struct(PNG_LIBPNG_VER_STRING, NULL, NULL, NULL);
  if (png_ptr == NULL) {
    fclose(fp);
  }

  info_ptr = png_create_info_struct(png_ptr);
  if (info_ptr == NULL) {
    png_destroy_write_struct(&png_ptr, &info_ptr);
  }

  /* Set up error handling. */
  if (setjmp(png_jmpbuf(png_ptr))) {
    png_destroy_write_struct(&png_ptr, &info_ptr);
  }

  /* Set image attributes. */
  png_set_IHDR(
    png_ptr,
    info_ptr,
    bitmap->width,
    bitmap->height,
    depth,
    PNG_COLOR_TYPE_RGB,
    PNG_INTERLACE_NONE,
    PNG_COMPRESSION_TYPE_DEFAULT,
    PNG_FILTER_TYPE_DEFAULT
  );

  /* Initialize rows of PNG. */
  row_pointers = (png_byte **)png_malloc(png_ptr, bitmap->height * sizeof(png_byte*));
  for (y = 0; y < bitmap->height; ++y) {
    png_byte* row =
      (png_byte *)png_malloc(png_ptr, bitmap->width * pixel_size * sizeof(uint8_t));
    row_pointers[y] = row;
    for (x = 0; x < bitmap->width; ++x) {
      Pixel* pixel = pixel_at(bitmap, x, y);
      *row++ = pixel->red;
      *row++ = pixel->green;
      *row++ = pixel->blue;
    }
  }

  /* Write the image data to "fp". */
  png_init_io(png_ptr, fp);
  png_set_rows(png_ptr, info_ptr, row_pointers);
  png_write_png(png_ptr, info_ptr, PNG_TRANSFORM_IDENTITY, NULL);

  /* The routine has successfully written the file, so we set
     "status" to a value which indicates success. */
  status = 0;

  for (y = 0; y < bitmap->height; ++y) {
    png_free(png_ptr, row_pointers[y]);
  }
  png_free(png_ptr, row_pointers);

  return status;
}

/* Given "value" and "max", the maximum value which we expect "value"
   to take, this returns an integer between 0 and 255 proportional to
   "value" divided by "max". */
__device__
int rgb_value(int value, int max) {
  return (int)(256 * (value / (double)max));
}

__device__
int iteration(complex<double> c, int limit = 1000) {
  int i = 0;
  double n;
  complex<double> z(0, 0);
  while ((n = norm(z)) < 4 && i < limit) {
    z = exp(z) - c;
    ++i;
  }

  // if (n < 4)
  //   return -1;
  // else
  return i;
}

__global__
void calc(Bitmap* bitmap) {
  // int index = blockIdx.x * blockDim.x + threadIdx.x;
  // int stride = gridDim.x * blockDim.x;

  // for (int i = index; i < n; i += stride) {
  //   result[i] = iteration(c[i]);
  // }

  double lowerX = 0, upperX = 4;
  double lowerY = -2, upperY = 2;

  int x, y;
  int xStride = gridDim.x * blockDim.x;
  int yStride = gridDim.y * blockDim.y;

  complex<double> t;
  int iter;
  Pixel* pixel;
  int width = bitmap->width;
  int height = bitmap->height;
  for (y = blockIdx.y * blockDim.y + threadIdx.y; y < height; y += yStride) {
    for (x = blockIdx.x * blockDim.x + threadIdx.x; x < width; x += xStride) {
      t = complex<double>(lowerX + (upperX - lowerX) * x / (width - 1),
                        lowerY + (upperY - lowerY) * y / (height - 1));
      iter = iteration(t);
      pixel = pixel_at(bitmap, x, y);
      pixel->red = rgb_value(1000 - iter, 1000);
      pixel->green = rgb_value(500 - iter, 1000);
      pixel->blue = rgb_value(200 - iter, 1000);
      // printf("(%d %d) %d (strides: %d %d) (w/h: %d %d) \n", x, y, iter, xStride, yStride, width, height);
    }
    // printf("(%d %d) %d \n", x, y, iter);
  }
}

int main() {
  Bitmap bitmap;
  bitmap.width = 200;
  bitmap.height = 200;
  hipMalloc(&bitmap.pixels, sizeof(Pixel) * bitmap.width * bitmap.height);

  Bitmap* devBitmap;
  hipMalloc(&devBitmap, sizeof(Bitmap));
  hipMemcpy(devBitmap, &bitmap, sizeof(Bitmap), hipMemcpyHostToDevice);

  dim3 threadsPerBlock(16, 16);
  dim3 numBlocks((bitmap.width + threadsPerBlock.x - 1) / threadsPerBlock.x,
                 (bitmap.height + threadsPerBlock.y - 1) / threadsPerBlock.y);

  calc<<<numBlocks, threadsPerBlock>>>(devBitmap);

  hipDeviceSynchronize();

  Pixel* devicePixels = bitmap.pixels;
  bitmap.pixels = new Pixel[bitmap.width * bitmap.height];
  hipMemcpy(bitmap.pixels, devicePixels, sizeof(Pixel) * bitmap.width * bitmap.height, hipMemcpyDeviceToHost);

  save_png_to_file(&bitmap, "fractal.png");

  hipFree(devicePixels);
  hipFree(devBitmap);

  delete[] bitmap.pixels;

  hipError_t error = hipGetLastError();
  std::cout << hipGetErrorString(error) << std::endl;

  return 0;
}
