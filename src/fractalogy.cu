
#include <hip/hip_runtime.h>
#include <png.h>
#include <unistd.h>
#include <stdio.h>
#include <stdlib.h>
#include <stdint.h>
#include <hip/hip_complex.h>
#include <iostream>
#include <cmath>

// A coloured pixel.
struct Pixel {
  uint8_t red;
  uint8_t green;
  uint8_t blue;
};

// A picture.
struct Bitmap {
  Pixel* pixels;
  size_t width;
  size_t height;
};

// Given "bitmap", this returns the pixel of bitmap at the point ("x", "y").
__host__ __device__
Pixel* pixel_at(const Bitmap& bitmap, int x, int y) {
  return bitmap.pixels + bitmap.width * y + x;
}

/* Write "bitmap" to a PNG file specified by "path"; returns 0 on
   success, non-zero on error. */
int save_png_to_file(Bitmap* bitmap, const char* path) {
  FILE* fp;
  png_structp png_ptr = NULL;
  png_infop info_ptr = NULL;
  size_t x, y;
  png_byte** row_pointers = NULL;

  /* "status" contains the return value of this function. At first
     it is set to a value which means 'failure'. When the routine
     has finished its work, it is set to a value which means
     'success'. */
  int status = -1;

  /* The following number is set by trial and error only. I cannot
     see where it is documented in the libpng manual. */
  int pixel_size = 3;
  int depth = 8;

  fp = fopen(path, "wb");
  if (!fp) {
    return status;
  }

  png_ptr = png_create_write_struct(PNG_LIBPNG_VER_STRING, NULL, NULL, NULL);
  if (png_ptr == NULL) {
    fclose(fp);
  }

  info_ptr = png_create_info_struct(png_ptr);
  if (info_ptr == NULL) {
    png_destroy_write_struct(&png_ptr, &info_ptr);
  }

  /* Set up error handling. */
  if (setjmp(png_jmpbuf(png_ptr))) {
    png_destroy_write_struct(&png_ptr, &info_ptr);
  }

  /* Set image attributes. */
  png_set_IHDR(
    png_ptr,
    info_ptr,
    bitmap->width,
    bitmap->height,
    depth,
    PNG_COLOR_TYPE_RGB,
    PNG_INTERLACE_NONE,
    PNG_COMPRESSION_TYPE_DEFAULT,
    PNG_FILTER_TYPE_DEFAULT
  );

  /* Initialize rows of PNG. */
  row_pointers = (png_byte **)png_malloc(png_ptr, bitmap->height * sizeof(png_byte*));
  for (y = 0; y < bitmap->height; ++y) {
    png_byte* row =
      (png_byte *)png_malloc(png_ptr, bitmap->width * pixel_size * sizeof(uint8_t));
    row_pointers[y] = row;
    for (x = 0; x < bitmap->width; ++x) {
      Pixel* pixel = pixel_at(*bitmap, x, y);
      *row++ = pixel->red;
      *row++ = pixel->green;
      *row++ = pixel->blue;
    }
  }

  /* Write the image data to "fp". */
  png_init_io(png_ptr, fp);
  png_set_rows(png_ptr, info_ptr, row_pointers);
  png_write_png(png_ptr, info_ptr, PNG_TRANSFORM_IDENTITY, NULL);

  /* The routine has successfully written the file, so we set
     "status" to a value which indicates success. */
  status = 0;

  for (y = 0; y < bitmap->height; ++y) {
    png_free(png_ptr, row_pointers[y]);
  }
  png_free(png_ptr, row_pointers);

  return status;
}

/* Given "value" and "max", the maximum value which we expect "value"
   to take, this returns an integer between 0 and 255 proportional to
   "value" divided by "max". */
__device__
int rgb_value(int value, int max) {
  return (int)(256 * (value / (double)max));
}

__device__
int iteration(hipDoubleComplex c, int limit = 1000) {
  int i = 0;
  double n;
  hipDoubleComplex z = make_hipDoubleComplex(0, 0);
  while ((n = hipCreal(hipCmul(hipConj(z), z))) < 4 && i < limit) {
    z = hipCadd(hipCmul(z, z), c);
    // z = exp(z) - c;
    // z = c * exp(-z) + z * z;
    // z = cuCadd(cuCmul(c, exp(cuCsub(make_cuDoubleComplex(0, 0), z))), cuCmul(z, z));
    ++i;
  }

  // if (n < 4)
  //   return -1;
  // else
  return i;
}

__global__
void calc(Bitmap bitmap) {
  // int index = blockIdx.x * blockDim.x + threadIdx.x;
  // int stride = gridDim.x * blockDim.x;

  // for (int i = index; i < n; i += stride) {
  //   result[i] = iteration(c[i]);
  // }

  double lowerX = -2, upperX = 2;
  double lowerY = -2, upperY = 2;

  int x, y;
  int xStride = gridDim.x * blockDim.x;
  int yStride = gridDim.y * blockDim.y;

  hipDoubleComplex t;
  int iter;
  Pixel* pixel;
  int width = bitmap.width;
  int height = bitmap.height;
  for (y = blockIdx.y * blockDim.y + threadIdx.y; y < height; y += yStride) {
    for (x = blockIdx.x * blockDim.x + threadIdx.x; x < width; x += xStride) {
      t = make_hipDoubleComplex(lowerX + (upperX - lowerX) * x / (width - 1),
                               lowerY + (upperY - lowerY) * y / (height - 1));
      iter = iteration(t);
      pixel = pixel_at(bitmap, x, y);
      pixel->red = rgb_value(1000 - iter, 1000);
      pixel->green = rgb_value(500 - iter, 1000);
      pixel->blue = rgb_value(200 - iter, 1000);
      // printf("(%d %d) %d (strides: %d %d) (w/h: %d %d) \n", x, y, iter, xStride, yStride, width, height);
    }
    // printf("(%d %d) %d \n", x, y, iter);
  }
}

int main(int argc, char** argv) {
  ////////////////////////////////////

  char *svalue = NULL, *rvalue = NULL, *tvalue = NULL, *filenameArg = NULL;
  int c;

  opterr = 0;

  while ((c = getopt (argc, argv, "s:r:t:o:")) != -1)
    switch (c)
      {
      case 's':
        svalue = optarg;
        break;
      case 'r':
        rvalue = optarg;
        break;
      case 't':
        tvalue = optarg;
        break;
      case 'o':
        filenameArg = optarg;
        break;
      case '?':
        if (optopt == 's')
          fprintf (stderr, "Option -%c requires an argument.\n", optopt);
        else if (isprint (optopt))
          fprintf (stderr, "Unknown option `-%c'.\n", optopt);
        else
          fprintf (stderr,
                   "Unknown option character `\\x%x'.\n",
                   optopt);
        return 1;
      default:
        abort ();
      }


  int width = 1000, height = 1000;
  double lowerX = -2, upperX = 2;
  double lowerY = -2, upperY = 2;
  int threads = 1;
  char filename[100] = "fractal.png";
  if (svalue != NULL)
    sscanf(svalue, "%dx%d", &width, &height);
  if (rvalue != NULL)
    sscanf(rvalue, "%lf:%lf:%lf:%lf", &lowerX, &upperX, &lowerY, &upperY);
  if (tvalue != NULL)
    sscanf(tvalue, "%d", &threads);
  if (filenameArg != NULL)
    sscanf(filenameArg, "%s", filename);
  printf ("svalue = %s;%dx%d\nrvalue = %s; %lf, %lf, %lf, %lf\n", svalue, width, height, rvalue, lowerX, upperX, lowerY, upperY);
  printf ("tvalue = %s; %d\n", tvalue, threads);
  printf ("filenameArg = %s; %s\n", filenameArg, filename);

  //////////////////////////////////
  Bitmap bitmap;
  bitmap.width = width;
  bitmap.height = height;

  size_t pixelsCount = bitmap.width * bitmap.height;
  size_t pixelsSize = pixelsCount * sizeof(Pixel);
  hipMalloc(&bitmap.pixels, pixelsSize);

  dim3 threadsPerBlock(16, 16);
  dim3 numBlocks((bitmap.width + threadsPerBlock.x - 1) / threadsPerBlock.x,
                 (bitmap.height + threadsPerBlock.y - 1) / threadsPerBlock.y);

  calc<<<numBlocks, threadsPerBlock>>>(bitmap);

  hipDeviceSynchronize();

  Pixel* devicePixels = bitmap.pixels;
  bitmap.pixels = new Pixel[pixelsCount];
  hipMemcpy(bitmap.pixels, devicePixels, pixelsSize, hipMemcpyDeviceToHost);

  save_png_to_file(&bitmap, filename);

  hipFree(devicePixels);
  delete[] bitmap.pixels;

  hipError_t error = hipGetLastError();
  std::cout << hipGetErrorString(error) << std::endl;

  return 0;
}
